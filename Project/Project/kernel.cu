#include "hip/hip_runtime.h"
/******************************************
*******************************************
***		Ariel Levin						***
***		ariel.lvn89@gmail.com			***
***		http://about.me/ariel.levin		***
*******************************************
******************************************/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>


hipError_t initCuda(float **cuda_a, float **cuda_b, float **cuda_emission, float a[], float b[], unsigned int num_of_states);
hipError_t emissionWithCuda(float emission[], float cuda_emission[], float cuda_a[], float cuda_b[], float obsrv, unsigned int num_of_states);
void freeCuda(float cuda_emission[], float cuda_a[], float cuda_b[]);

bool	WITH_LOGS;


__global__ void emissionKernel(float emission[], float a[], float b[], float obsrv, int N, bool withLog)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N)
	{
		if (withLog)
			emission[i] = log(a[i] * exp(-pow(obsrv - b[i], 2)));
		else
			emission[i] = a[i] * exp(-pow(obsrv - b[i], 2));
	}
}


// Helper function for using CUDA to calculate the emission function
hipError_t emissionWithCuda(float emission[], float cuda_emission[], float cuda_a[], float cuda_b[], float obsrv, unsigned int num_of_states)
{
	hipError_t cudaStatus;

	// Invoke kernel 
	int threadsPerBlock = 1024;
	int blocksPerGrid = (num_of_states + threadsPerBlock - 1) / threadsPerBlock;
	emissionKernel << < blocksPerGrid, threadsPerBlock >> >(cuda_emission, cuda_a, cuda_b, obsrv, num_of_states, WITH_LOGS);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "emissionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeCuda(cuda_emission, cuda_a, cuda_b);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeCuda(cuda_emission, cuda_a, cuda_b);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(emission, cuda_emission, num_of_states * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeCuda(cuda_emission, cuda_a, cuda_b);
		return cudaStatus;
	}

	return cudaStatus;
}

hipError_t initCuda(float **cuda_a, float **cuda_b, float **cuda_emission, float a[], float b[],
	unsigned int num_of_states, bool withLog)
{
	hipError_t cudaStatus;
	*cuda_a = 0;
	*cuda_b = 0;
	*cuda_emission = 0;

	WITH_LOGS = withLog;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freeCuda(*cuda_emission, *cuda_a, *cuda_b);
		return cudaStatus;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)cuda_a, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeCuda(*cuda_emission, *cuda_a, *cuda_b);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)cuda_b, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeCuda(*cuda_emission, *cuda_a, *cuda_b);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)cuda_emission, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeCuda(*cuda_emission, *cuda_a, *cuda_b);
		return cudaStatus;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(*cuda_a, a, num_of_states * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeCuda(*cuda_emission, *cuda_a, *cuda_b);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(*cuda_b, b, num_of_states * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeCuda(*cuda_emission, *cuda_a, *cuda_b);
		return cudaStatus;
	}

	return cudaStatus;
}

void freeCuda(float cuda_emission[], float cuda_a[], float cuda_b[])
{
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_emission);
}

