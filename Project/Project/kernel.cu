#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>


hipError_t initCuda(float **cuda_a, float **cuda_b, float **cuda_emission, float a[], float b[], unsigned int num_of_states);
hipError_t emissionWithCuda(float emission[], float cuda_emission[], float cuda_a[], float cuda_b[], float obsrv, unsigned int num_of_states);
void freeCuda(float cuda_emission[], float cuda_a[], float cuda_b[]);


__global__ void emissionKernel(float emission[], float a[], float b[], float obsrv)
{
	int i = threadIdx.x;
	emission[i] = a[i] * exp(-pow(obsrv - b[i], 2));
}


hipError_t initCuda(float **cuda_a, float **cuda_b, float **cuda_emission, float a[], float b[], unsigned int num_of_states)
{
	float *tmp_a = 0;
	float *tmp_b = 0;
	float *tmp_emission = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		freeCuda(tmp_emission, tmp_a, tmp_b);
		return cudaStatus;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&tmp_a, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeCuda(tmp_emission, tmp_a, tmp_b);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&tmp_b, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeCuda(tmp_emission, tmp_a, tmp_b);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&tmp_emission, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		freeCuda(tmp_emission, tmp_a, tmp_b);
		return cudaStatus;
	}



	// Copy input vectors from host memory to GPU buffers.

	cudaStatus = hipMemcpy(tmp_a, a, num_of_states * sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cuda_a), a, num_of_states * sizeof(float), size_t(0), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeCuda(tmp_emission, tmp_a, tmp_b);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(tmp_b, b, num_of_states * sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(cuda_b), b, num_of_states * sizeof(float), size_t(0), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeCuda(tmp_emission, tmp_a, tmp_b);
		return cudaStatus;
	}

	*cuda_a = tmp_a;
	*cuda_b = tmp_b;
	*cuda_emission = tmp_emission;

	return cudaStatus;
}

// Helper function for using CUDA to calculate the emission function
hipError_t emissionWithCuda(float emission[], float cuda_emission[], float cuda_a[], float cuda_b[], float obsrv, unsigned int num_of_states)
{
	hipError_t cudaStatus;

	// Launch a kernel on the GPU with one thread for each element.
	emissionKernel << < 1, num_of_states >> >(cuda_emission, cuda_a, cuda_b, obsrv);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "emissionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		freeCuda(cuda_emission, cuda_a, cuda_b);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		freeCuda(cuda_emission, cuda_a, cuda_b);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(emission, cuda_emission, num_of_states * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		freeCuda(cuda_emission, cuda_a, cuda_b);
		return cudaStatus;
	}

	return cudaStatus;
}

void freeCuda(float cuda_emission[], float cuda_a[], float cuda_b[])
{
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_emission);
}

