#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>


hipError_t emissionWithCuda(float emission[], float a[], float b[], float obsrv, int num_of_states);


//float *dev_a, *dev_b, *dev_obsrv;


__global__ void emissionKernel(float emission[], float a[], float b[], float obsrv[])
{
	int i = threadIdx.x;
	emission[i] = a[i] * exp(-pow(obsrv[0] - b[i], 2));
}

// Helper function for using CUDA to calculate the emission function
hipError_t emissionWithCuda(float emission[], float a[], float b[], float obsrv, int num_of_states)
{
	float *dev_emission = 0;
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_obsrv = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_emission, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, num_of_states * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_obsrv, sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, num_of_states * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, num_of_states * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_obsrv, &obsrv, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	emissionKernel << < 1, num_of_states >> >(dev_emission, dev_a, dev_b, dev_obsrv);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "emissionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(emission, dev_emission, num_of_states * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_emission);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_obsrv);

	return cudaStatus;
}

